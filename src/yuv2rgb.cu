#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_runtime.h>
#include "yuv2rgb.cuh"

__device__ inline float clamp(float val, float mn, float mx)
{
	return (val >= mn)? ((val <= mx)? val : mx) : mn;
}

__global__ void gpuConvertYUYVtoBGR_kernel(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx*2 >= width) {
		return;
	}

	for (int i = 0; i < height; ++i) {
		int y0 = src[i*width*2+idx*4+0];
		int cb = src[i*width*2+idx*4+1];
		int y1 = src[i*width*2+idx*4+2];
		int cr = src[i*width*2+idx*4+3];

		//r0
		dst[i*width*3+idx*6+2] = clamp(1.164f * (y0 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		//g0
		dst[i*width*3+idx*6+1] = clamp(1.164f * (y0 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		//b0
		dst[i*width*3+idx*6+0] = clamp(1.164f * (y0 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);

		//r0
		dst[i*width*3+idx*6+5] = clamp(1.164f * (y1 - 16) + 1.596f * (cr - 128)                      , 0.0f, 255.0f);
		//g1
		dst[i*width*3+idx*6+4] = clamp(1.164f * (y1 - 16) - 0.813f * (cr - 128) - 0.391f * (cb - 128), 0.0f, 255.0f);
		//b1
		dst[i*width*3+idx*6+3] = clamp(1.164f * (y1 - 16)                       + 2.018f * (cb - 128), 0.0f, 255.0f);
	}
}


void gpuConvertYUYVtoBGR(unsigned char *src, unsigned char *dst,
		unsigned int width, unsigned int height)
{
	static unsigned int blockSize = 1024;
	static unsigned int numBlocks = (width / 2 + blockSize - 1) / blockSize;
	gpuConvertYUYVtoBGR_kernel<<<numBlocks, blockSize>>>(src, dst, width, height);
	hipDeviceSynchronize();
}
